#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"

#define DOT(X, Y) (X[0]*Y[0] + X[1]*Y[1] + X[2]*Y[2])

__device__ const float L[] = {4, 4, -1};       /* Light source */
__device__ const float C[] = {0, 12, 0};       /* Sphere center */
// __device__ const float R = 6;                  /* Sphere radius */
__device__ const float Wy = 2;                 /* Window y-position */
__device__ const float Wmax = 2;               /* Window dimensions */
__device__ const float magicnum = 36 - 144;

// __device__ inline float SIN(float x) {
//     float xshift = x - M_PI/2;
//     return 1 - xshift*xshift/2 + xshift*xshift*xshift*xshift/24;
// }

// __device__ inline float COS(float x) {
//     float xshift = x - M_PI/2;
//     return -xshift + xshift*xshift*xshift/6 - xshift*xshift*xshift*xshift*xshift/120;
// }

__host__ void writeOutput(void *data, long sz){
    printf("Writing file: sphere.bin\n");
    FILE *file = fopen("sphere.bin", "w");
    fwrite(data, sizeof(float), sz, file);
    fclose(file);
}

__device__ inline void prod(float a, float *X) {
    for (int i = 0; i < 3; i++)
        X[i] *= a;
}

__global__ void simulate_rays(float *Gdev, int *samplesdev, int nrays, int ngrid, int nt) {
    int row, col, samplespriv = 0;
    float V[3], W[3], I[3], N[3], S[3];
    float phi, costheta, sintheta, t, VdotC, disc, NdotS, b;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateXORWOW_t rng;
    hiprand_init(id, id, 0, &rng);

    for (int z = id; z < nrays; z += nt) {
        do {
            phi = hiprand_uniform(&rng) * M_PI;
            costheta = hiprand_uniform(&rng) * 2 - 1;
            sintheta = sqrt(1 - costheta*costheta);
            samplespriv += 2;

            V[0] = sintheta * cos(phi);
            V[1] = sintheta * sin(phi);
            V[2] = costheta;

            for (int i = 0; i < 3; i++) {
                W[i] = (Wy / V[1]) * V[i];
            }

            VdotC = DOT(V, C);

        } while ((disc = VdotC*VdotC + magicnum) < 0 || -Wmax >= W[0] || W[0] >= Wmax || -Wmax >= W[2] || W[2] >= Wmax);

        t = VdotC - sqrt(disc);
        for (int i = 0; i < 3; i++)
            I[i] = t * V[i];
        for (int i = 0; i < 3; i++) {
            N[i] = I[i] - C[i];
            S[i] = L[i] - I[i];
        }
        prod(1/sqrt(DOT(N, N)), N);
        prod(1/sqrt(DOT(S, S)), S);
        NdotS = DOT(N, S);
        b = NdotS > 0 ? NdotS : 0;

        row = ngrid - 1 - floorf((W[0] + Wmax) * ngrid / (2 * Wmax));
        col = floorf((W[2] + Wmax) * ngrid / (2 * Wmax));

        atomicAdd(&Gdev[row*ngrid + col], b);
    }
    atomicAdd(&samplesdev[blockIdx.x], samplespriv);

    return;
}

int main(int argc, char **argv) {
    long long total_samples = 0;
    GpuTimer timer, ktimer;
    int nrays, ngrid, nblocks, ntpb, nt;

    timer.Start();

    if (argc < 5) {
        printf("Usage: ./ray_tracing_gpu <nrays> <ngrid> <nblocks> <ntpb>\n");
        exit(0);
    } else {
        nrays = atoi(argv[1]);
        ngrid = atoi(argv[2]);
        nblocks = atoi(argv[3]);
        ntpb = atoi(argv[4]);
    }

    printf("Parameters: nrays: %d, ngrid: %d, nblocks: %d, threads per block: %d\n", nrays, ngrid, nblocks, ntpb);

    nt = nblocks * ntpb;

    float *G = (float *)malloc(sizeof(float)*ngrid*ngrid);
    float *Gdev;
    hipMalloc((void **)&Gdev, ngrid*ngrid*sizeof(float));
    hipMemset((void *)Gdev, 0, ngrid*ngrid*sizeof(float));

    int *samples = (int *)malloc(sizeof(int)*nblocks);
    int *samplesdev;
    hipMalloc((void **)&samplesdev, sizeof(int)*(nblocks));
    hipMemset((void *)samplesdev, 0, sizeof(int)*(nblocks));

    ktimer.Start();
    simulate_rays<<<nblocks, ntpb, ntpb>>>(Gdev, samplesdev, nrays, ngrid, nt);
    ktimer.Stop();

    hipMemcpy(G, Gdev, ngrid*ngrid*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(samples, samplesdev, sizeof(int)*(nblocks), hipMemcpyDeviceToHost);
    writeOutput(G, ngrid*ngrid);
    for (int i = 0; i < nblocks; i++) {
        total_samples += samples[i];
    }

    printf("RNG samples: %ld\n", total_samples);
    printf("Kernel time = %g ms\n", ktimer.Elapsed());

    free(G);
    free(samples);
    hipFree(samplesdev);
    hipFree(Gdev);

    timer.Stop();
    printf("Total time = %g ms\n", timer.Elapsed());
    return 0;
}